#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "kernel.h"
#include "device_cuda.h"

extern hipblasHandle_t handle;

#ifdef CUBLAS
void kernel (unsigned int n, double* a, const double* b, const double* c)
{
    int size = n * n * sizeof(double);

    double* d_a;
    double* d_b;
    double* d_c;
	CHECK(hipMalloc(&d_a, size));
    CHECK(hipMalloc(&d_b, size));
    CHECK(hipMalloc(&d_c, size));

    CHECK(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));

    dim3 nbThreads (n, n);
    dim3 nbBlocks (1,1);
    
    if ( n > 32 )
    {
        nbThreads.x = 32;
        nbThreads.y = 32;
        nbBlocks.x = ceil(double(n)/double(nbThreads.x));
        nbBlocks.y = ceil(double(n)/double(nbThreads.y));
    }
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_b, n, d_c, n, &beta, d_a, n);
        
	CHECK(hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}
#endif


#ifdef CUBLAS_WO_DT
void kernel (unsigned int n, double* a, const double* b, const double* c)
{ 
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, b, n, c, n, &beta, a, n);
}
#endif