#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "kernel.h"
#include "device_cuda.h"

extern hipblasHandle_t handle;

#ifdef CUBLAS
void kernel (unsigned int n, double* a, const double* b, const double* c)
{
    int size = n * n * sizeof(double);

    double* d_a;
    double* d_b;
    double* d_c;
	CHECK(hipMalloc(&d_a, size));
    CHECK(hipMalloc(&d_b, size));
    CHECK(hipMalloc(&d_c, size));

    CHECK(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));

    double alpha = 1.0f;
    double beta = 0.0f;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_c, n, d_b, n, &beta, d_a, n);
        
	CHECK(hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}
#endif


#ifdef CUBLAS_WO_DT
void kernel (unsigned int n, double* a, const double* b, const double* c)
{ 
    double alpha = 1.0f;
    double beta = 0.0f;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, c, n, b, n, &beta, a, n);
}
#endif