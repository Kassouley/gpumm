#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "device_cuda.h"

__global__ void kernel_cuda (unsigned int n, double* a, const double* b, const double* c)
{ 
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) 
    {
        double value = 0.0;
        for (unsigned int i = 0; i < n; i++) 
        {
            value += b[row * n + i] * c[i * n + col];
        }
        a[row * n + col] = value;
    }
}

#ifdef CUDA_WO_DT
void kernel (unsigned int n, double* a, const double* b, const double* c)
{
    dim3 nbThreads (n, n);
    dim3 nbBlocks (1,1);
    if ( n > 32 )
    {
        nbThreads.x = 32;
        nbThreads.y = 32;
        nbBlocks.x = ceil(double(n)/double(nbThreads.x));
        nbBlocks.y = ceil(double(n)/double(nbThreads.y));
    }
    kernel_cuda<<<nbBlocks, nbThreads>>>(n, a, b, c);
}
#endif

#ifdef CUDA
void kernel (unsigned int n, double* a, const double* b, const double* c)
{
    int size = n * n * sizeof(double);

    double* d_a;
    double* d_b;
    double* d_c;
    
	CHECK(hipMalloc(&d_a, size));
    CHECK(hipMalloc(&d_b, size));
    CHECK(hipMalloc(&d_c, size));

    CHECK(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_c, c, size, hipMemcpyHostToDevice));

    dim3 nbThreads (n, n);
    dim3 nbBlocks (1,1);
    if ( n > 32 )
    {
        nbThreads.x = 32;
        nbThreads.y = 32;
        nbBlocks.x = ceil(double(n)/double(nbThreads.x));
        nbBlocks.y = ceil(double(n)/double(nbThreads.y));
    }

    kernel_cuda<<<nbBlocks, nbThreads>>>(n, d_a, d_b, d_c);
        
	CHECK(hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
}
#endif